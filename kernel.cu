
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
//__global__ prefixes a kernel the following function will run in parallel on the GPU(device)
//runs on the device
//is called from host code


__global__ void vecAddKernel(float* A, float* B, float* C, int n) {

    // Calculate global thread index based on the block and thread indices ----
  
    //INSERT KERNEL CODE HERE
      int i =  threadIdx.x + blockIdx.x * blockDim.x;
      if(i < n)
      {
        C[i] = A[i] + B[i];
      }



    // Use global index to determine which elements to read, add, and write ---

    //INSERT KERNEL CODE HERE
   //waits until all threads within the same block has reached the command and all threads within a wrap
   //one can specify synchronization points in the kernel by calling the __syncthreads()
   __syncthreads();



}

